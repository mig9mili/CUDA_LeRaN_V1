
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void vectoradd(float* a , float *b , float *res , int n ){

 // get the  gloable thread ID 
 int id = blockIdx.x * blockDim.x + threadIdx.x;

 if(id < n){
    res[id] = a[id] + b[id] ;
 }
}

int main(){
     
     int n = 1000;
     size_t size = n* sizeof(float);

     //host vector
     float *h_a , *h_b , *h_result;

     //devic vetor 
     float  *d_a , *d_b , *d_result;

     //allocate memory for host vector
     
     h_a = (float*)malloc(size);
     h_b = (float*)malloc(size);
     h_result = (float*)malloc(size);

     //intializ the host vector

     for(int i =0; i < n ; i++){
        h_a[i] = rand() /(float)RAND_MAX ;
        h_b[i] = rand() /(float)RAND_MAX ;
     }

     //allocate memory for device vectors
     hipMalloc(&d_a , size);
     hipMalloc(&d_b, size);
     hipMalloc(&d_result , size);

     //copy input to device 

     hipMemcpy(d_a , h_a ,size , hipMemcpyHostToDevice);
     hipMemcpy(d_b, h_b ,size , hipMemcpyHostToDevice);

     int threadsPerblock = 256 ;

    int blocksPerGrid = (n + threadsPerblock -1) / threadsPerblock ;

    vectoradd<<<blocksPerGrid , threadsPerblock >>>(d_a , d_b , d_result , n);
    

    hipMemcpy(h_result,d_result, size, hipMemcpyDeviceToHost);

    for(int i =0 ; i <5 ; i++){
        printf("%.2f + %.2f = %.2f \n " , h_a[i],h_b[i],h_result[i] );
    }
  


hipFree(d_a);
hipFree(d_b);
hipFree(d_result);

free(h_a);
free(h_b);
free(h_result);

return 0;
}